
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
__global__ void multiply(int *a,int *b,int *c,int n,int m)
{

int row = blockIdx.y*blockDim.y+threadIdx.y;
int col = blockIdx.x*blockDim.x+threadIdx.x;
int result =0;
if(row<n&&col<m)
{
for(int i =0;i<n;i++)
{
result+=a[row*n+i]*b[i*m+col];
}
c[row*m+col]=result;
}

}
int main()
{

int *a,*b,*c;
int n=3;
int m=3;
int size = n*sizeof(int);
a = (int*)malloc(size*size);
b = (int*)malloc(size*size);
c = (int*)malloc(size*size);

for(int i=0;i<n;i++)
{
for(int j=0;j<n;j++)
{
*(a+i*n+j)=i*n+(j+1);
*(b+i*n+j)=i*n+(j+1);
}
}

printf("\\nThe Matrix a is:\\n");
for(int i=0;i<n;i++)
{
for(int j=0;j<n;j++)
{
printf("%d\t",*(a+i*n+j));
}
printf("\\n");
}

printf("\\nThe Matrix b is:\\n");
for(int i=0;i<n;i++)
{
for(int j=0;j<n;j++)
{
printf("%d\t",*(b+i*n+j));
}


printf("\\n");
}

int *d_a,*d_b,*d_c;
hipMalloc(&d_a,size*size);

hipMalloc(&d_b,size*size);

hipMalloc(&d_c,size*size);
hipMemcpy(d_a,a,size*size,hipMemcpyHostToDevice);

hipMemcpy(d_b,b,size*size,hipMemcpyHostToDevice);
dim3 dimGrid(1,1);
dim3 dimBlock(16,16);
multiply<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,n,m);

hipMemcpy(c,d_c,size*size,hipMemcpyDeviceToHost);

printf("\\nThe Matrix c is:\\n");
for(int i=0;i<n;i++)
{
for(int j=0;j<n;j++)
{
printf("%d\t",*(c+i*n+j));
}


printf("\\n");
}

hipFree(d_a);

hipFree(d_b);

hipFree(d_c);





return 0;

}
